#include "hip/hip_runtime.h"
// 0/128 -> 146.263 sec / 461.514 sec
// 1/128 -> 479.29  sec / 1440.94  sec
// 2/128 -> 150.118 sec / 453.605 sec

// 0/64 -> 87.2188 sec / 263.548 sec
// 1/64 -> 153.032 sec / 461.115 sec
// 2/64 -> 79.6683 sec / 240.896 sec

// 0/48 -> 35.8488 sec / 109.224 sec
// 1/48 -> 58.8207 sec / 180.714  sec
// 2/48 -> 12.7201 sec / 39.7368 sec

// 0/32 -> 26.3703  sec / 80.398 sec
// 1/32 -> 23.4618 sec / 71.6661 sec
// 2/32 -> 8.22882 sec / 25.626 sec

// 0/16 -> 7.27655 sec / 22.9368 sec
// 1/16 -> 4.11855 sec / 13.4249 sec
// 2/16 -> 2.26796 sec / 7.72694 sec

// 0/8 -> 1.65296  sec / 6.06111 sec
// 1/8 -> 1.10638 sec /  4.07978 sec
// 2/8 -> 0.776116 sec / 2.95669sec
#define use 1
#define clusterDimension 64

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <math.h>
#include <time.h>

#define iterations 3
#define FLOAT_MAX 1e+37
#define numPoints 1024*1024
#define numClusters 1024*3
#define ConstantMemFloats (64*1024)/4			//	64KB/4
#define rand_range 100
#define PC 0
#if PC == 1
/*Works on Windows!*/
double microtime() { return (double)time(NULL); }
#else
/*Woks on Linux*/
#include <sys/time.h>
double microtime(void)
{
	struct timeval t;
	gettimeofday(&t, 0);
	return 1.0e6*t.tv_sec + (double)t.tv_usec;
}
#endif



__constant__ float d_cons_centers[ConstantMemFloats];

__host__ void printDeviceInfo(){
	FILE * fp;
	fp = fopen("specifications.txt", "w");
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		fprintf(fp, "Name = %s\n", prop.name);
		fprintf(fp, "Number of Multi-processors = %d\n", (int)prop.multiProcessorCount);
		fprintf(fp, "Max threads per Block = %d\n", (int)prop.maxThreadsPerBlock);
		fprintf(fp, "Max threads per SM = %d\n", (int)prop.maxThreadsPerMultiProcessor);

		fprintf(fp, "Global Memory = %u B\n", (int)prop.totalGlobalMem);
		fprintf(fp, "L2 SIZE = %d bytes\n", (int)prop.l2CacheSize);
		fprintf(fp, "Shared memory per SM = %d B\n", (int)prop.sharedMemPerBlock);
		fprintf(fp, "Total Constant Memory = %d B\n", (int)prop.totalConstMem);

		fprintf(fp, "Registers per SM = %d\n", (int)prop.regsPerBlock);
		fprintf(fp, "Average Registers per Thread = %d\n", (int)prop.regsPerBlock / (int)prop.maxThreadsPerMultiProcessor);
		fprintf(fp, "Warp size = %d threads\n", (int)prop.warpSize);
		

		fprintf(fp, "Version = %d.%d\n", (int)prop.major , (int)prop.minor);
	}

	fclose(fp);
}

__host__ void generate_random_points(float *h_points){

	//Randomly generating points using rand()
	srand((unsigned int)time(0));
	for (int i = 0; i < numPoints; i++){
		for (int j = 0; j < clusterDimension; j++)
		{
			h_points[i*clusterDimension + j] = (float)(rand() % rand_range);
		}
	}

}
__host__ void generate_random_centers(float *h_points, float *h_centers_old, float *h_centers_new){
	//Selecting random points using Floyd's Algorithm
	int *rand_idx = (int *)malloc(numClusters*sizeof(int));
	int *flag = (int *)malloc(numPoints*sizeof(int));

	memset(rand_idx, 0, numClusters*sizeof(int));
	memset(flag, 0, numPoints*sizeof(int));

	int n = numPoints;
	int m = 0;
	for (n = (numPoints - numClusters); n < numPoints && m < numClusters; n++, m++){
		int r = rand() % (n + 1);

		if (flag[r] == 1){
			/*Works since previous iteration had rand() % n , and thus
			 it is not possible that n was chosen!*/
			r = n;
		}
		rand_idx[m] = r;
		flag[r] = 1;
	}


	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++)
		{
			h_centers_old[i*clusterDimension + j] = h_points[rand_idx[i] * clusterDimension + j];
			h_centers_new[i*clusterDimension + j] = h_points[rand_idx[i] * clusterDimension + j];
		}
	}

}
__host__ float calculate_norm(float *h_centers_old, float *h_centers_new){
	float diff_norm = 0.0f;
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			diff_norm += fabsf(h_centers_old[i*clusterDimension + j] - h_centers_new[i*clusterDimension + j]);
		}
	}

	return diff_norm;
}

__host__ void member_division(float *h_centers_new, int *h_memberCounter){
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			h_centers_new[i*clusterDimension + j] /= h_memberCounter[i];
		}
	}

}

__host__ void copy_centers(float * h_centers_old, float* h_centers_new){
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			h_centers_old[i*clusterDimension + j] = h_centers_new[i*clusterDimension + j];

		}
	}
}



//Costs 500 µ seconds! do not use!
__device__ float distance_func(float *point1, float *point2){
	float distance = 0.0f;
	for (int k = 0; k < clusterDimension; k++){
		distance += sqrtf((point1[k] - point2[k]) * (point1[k] - point2[k]));
	}
	return distance;

}

__global__ void calc_distance0(float *d_centers, float *d_points, int *d_clusterIdx, float *d_mindistances){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int min_pos = -1;
	if (i < numPoints){
		float min_dist = d_mindistances[i];
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[i*clusterDimension + j];
		}


		for (int k = 0; k < numClusters; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(points[j] - d_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}

		d_mindistances[i] = min_dist;
		d_clusterIdx[i] = min_pos;
	}
}

__global__ void calc_distance1(float *d_points, int *d_clusterIdx, float *d_mindistances, int step, int num_copy, int max_cached){

	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int tx = threadIdx.x;
	int min_pos = -1;
	extern __shared__ float s_points[];

	/*Getting the value from previous iterations*/
	if (i < numPoints){
		float min_dist = d_mindistances[i];
		float old_min_dist = d_mindistances[i];

		for (int j = 0; j < clusterDimension; j++){
			s_points[tx*clusterDimension + j] = d_points[i*clusterDimension + j];
		}

		for (int k = 0; k < num_copy; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(s_points[tx*clusterDimension + j] - d_cons_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}

		/*Only update if there were changes!!*/
		if (min_dist < old_min_dist){
			d_mindistances[i] = min_dist;
			d_clusterIdx[i] = step*max_cached + min_pos;
		}
	}
}


/*
Hoping that the point is in L1 cache!
Need to try this with 48KB L1 cache!
Probably saves a lot of index calculations and array accesses
Probably everything is in registers
*/
__global__ void calc_distance2(float *d_points, int *d_clusterIdx, float *d_mindistances, int step, int num_copy, int max_cached){

	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int min_pos = -1;

	/*Getting the value from previous iterations*/
	if (i < numPoints){
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[i*clusterDimension + j];
		}
		float min_dist = d_mindistances[i];
		float old_min_dist = d_mindistances[i];

		for (int k = 0; k < num_copy; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(points[j] - d_cons_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}

		/*Only update if there were changes!!*/
		if (min_dist < old_min_dist){
			d_mindistances[i] = min_dist;
			d_clusterIdx[i] = step*max_cached + min_pos;
		}
	}
}


__global__ void generate_new_center0(float *d_points, float *d_centers, int *d_clusterIdx, int * d_member_counter){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		for (int j = 0; j < clusterDimension; j++){
			atomicAdd(&d_centers[clusterDimension*clusterId + j], d_points[i*clusterDimension + j]);
		}
		atomicAdd(&d_member_counter[clusterId], 1);
	}

}

__global__ void generate_new_center1(float *d_points, float *d_centers, int *d_clusterIdx, int *d_member_counter, int split_steps, int split_size){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int tx = threadIdx.x;
	extern __shared__ float s_centers[];
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[i*clusterDimension + j];
		}

		for (int j = 0; j < split_steps; j++){
			/*
			Example....
			3*3 -> 9 to 11
			4*3 -> 12 to 14
			5*3 -> 15 to 17
			start -> 13 , 14 , 15 , 16
			end   -> 15 , 16 , 17 , 18
			diff  ->  3  , 2  , 1  , 0
			*/
			int max_length;
			int end_point = (j + 1)*split_size - 1;
			if (end_point <= clusterDimension - 1){
				max_length = split_size;
			}
			else{
				max_length = clusterDimension - j*split_size;
			}

			/*Collabaratively Load 0's in shared memory!*/
			/*Each thread 0's out ele_to_zero entries*/
			int total_ele = (48 * 1024) / (4);
			int ele_to_zero = (int)ceil(total_ele *1.0f / blockDim.x);

			for (int k = tx*ele_to_zero; k < (tx + 1)*ele_to_zero; k++){
				if (k < total_ele)
					s_centers[k] = 0;
			}
			__syncthreads();

			for (int k = 0; k < max_length; k++){
				atomicAdd(&s_centers[max_length*clusterId + k], points[j*split_size + k]);
			}
			__syncthreads();


			/*Collabaratively write back to d_centers!*/
			/*Not using all the threads . Need to find a smarter way to do this!*/
			int clus = ceil(1.0f*numClusters / blockDim.x);
			for (int l = tx*clus; l < (tx + 1)*clus; l++){
				if (l < numClusters)
					for (int k = 0; k < max_length; k++){
						atomicAdd(&d_centers[j*split_size + clusterDimension*l + k], s_centers[max_length*l + k]);
					}
			}
			__syncthreads();

		}
		atomicAdd(&d_member_counter[clusterId], 1);
	}
}

__global__ void squaredist(float *d_mindistances){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < numPoints){
		d_mindistances[i] *= d_mindistances[i];
	}
}


int main(int argc, char **argv){

	printDeviceInfo();
	double clk1, clk2, mclk1, mclk2, kmeansclk1, kmeansclk2;
	mclk1 = microtime();

	/*For maintaining multiple iterations of kmeans*/
	/*Usese RSS(Residual sum of squares) to determine best iteration*/
	/*RSS_score = sum((di)^2) for all di , where di is distance from cluster center*/
	double RSS_score = 0.0;
	double minRSS_score = (float)FLOAT_MAX;
	int *h_min_RSS_clusterIdx = (int *)malloc(numPoints*sizeof(int));

	/*For carrying out one iteration of Kmeans*/
	float *d_points, *d_centers, *d_mindistances;
	int *d_clusterIdx, *d_member_counter;
	float diff_norm = (float)FLOAT_MAX;

	//Calculations for calc_distance()
	/*Block fitting*/
	int ThreadsPerBlock = 1024;
	if ((48 * 1024) / (clusterDimension*sizeof(float)) < 1024)
		ThreadsPerBlock = (48 * 1024) / (clusterDimension*sizeof(float));
	int NumBlocks = (int)ceil(numPoints*1.0f / ThreadsPerBlock);

	/*Shared memory max fitting*/
	int smem_size = ThreadsPerBlock*clusterDimension*sizeof(float);

	/*Constant memory max fitting*/
	int max_cached, distance_steps;
	if (numClusters*clusterDimension * 4.0 <= 64 * 1024 * 1.0f){
		/*I can fit all the centers!*/
		max_cached = numClusters;
		distance_steps = 1;
	}
	else{
		/*One Point takes clusterDimension*4 memory , how mant can I fit in 64k?*/
		max_cached = (int)floor(64 * 1024 * 1.0f / (clusterDimension * 4));
		distance_steps = (int)ceil(numClusters*1.0f / max_cached*1.0f);
	}
	printf("Calculations for max_distance() function!\n");
	printf("numClusters = %d \tThreadsPerBlock = %d \t NumBlocks = %d\t smem_size = %d\n", numClusters, ThreadsPerBlock, NumBlocks, smem_size);
	printf("max_cached %d\t distance_steps %d\n\n", max_cached, distance_steps);

	//Calculations for generate_new_centers()
	int split_size = (int)floor((48 * 1024) / (numClusters * 4.0f));
	int split_steps = (int)ceil(clusterDimension *1.0f / split_size);
	printf("Calculations for generate_new_centers() function!\n");
	printf("split_size = %d \t split_steps = %d \n\n", split_size, split_steps);

	/*Stores the points*/
	float *h_points = (float *)malloc(clusterDimension*numPoints*sizeof(float));

	/*Need two arrays one for old centers, and one for new  for calculating NORM*/
	float *h_centers_old = (float *)malloc(clusterDimension*numClusters*sizeof(float));
	float *h_centers_new = (float *)malloc(clusterDimension*numClusters*sizeof(float));
	float *h_centers_zero = (float *)malloc(clusterDimension*numClusters*sizeof(float));

	/*Stores cluster indexes of all the points*/
	float *h_mindistances = (float *)malloc(numPoints*sizeof(float));
	int *h_clusterIdx = (int *)malloc(numPoints*sizeof(int));

	/*Consists number of members in a clusters*/
	int *h_member_counter = (int *)malloc(numClusters*sizeof(int));

	hipMalloc((void**)& d_points, clusterDimension*numPoints*sizeof(float));
	hipMalloc((void**)& d_centers, clusterDimension*numClusters*sizeof(float));
	hipMalloc((void**)& d_mindistances, numPoints*sizeof(float));
	hipMalloc((void **)&d_clusterIdx, numPoints*sizeof(int));
	hipMalloc((void **)&d_member_counter, numClusters*sizeof(int));
	generate_random_points(h_points);


	hipMemcpy(d_points, h_points, clusterDimension*numPoints*sizeof(float), hipMemcpyHostToDevice);
	memset(h_centers_zero, 0, clusterDimension*numClusters*sizeof(float));
	hipDeviceSynchronize();

	int count = 0;
	while (count < iterations){
		generate_random_centers(h_points, h_centers_old, h_centers_new);
		/*Each co-ordinate has a change less than 0.001 on average!*/
		kmeansclk1 = microtime();
		while (diff_norm > (numPoints*clusterDimension) / 1000.0){
			clk1 = microtime();

			for (int i = 0; i < numPoints; i++){
				h_clusterIdx[i] = INT_MAX;
				h_mindistances[i] = (float)FLOAT_MAX;
			}
			memset(h_member_counter, 0, numClusters*sizeof(int));
			hipMemcpy(d_clusterIdx, h_clusterIdx, numPoints*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(d_mindistances, h_mindistances, numPoints*sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_member_counter, h_member_counter, numClusters*sizeof(int), hipMemcpyHostToDevice);

			hipDeviceSynchronize();
			clk2 = microtime();
			printf("\nPART 1 :Iteration = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

			clk1 = microtime();
#if use == 0
			hipMemcpy(d_centers, h_centers_new, clusterDimension*numClusters*sizeof(float), hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			calc_distance0 << <(int)ceil(numPoints / 1024.0), 1024 >> >(d_centers, d_points, d_clusterIdx, d_mindistances);
			clk2 = microtime();
#elif use == 1
			for (int step = 0; step < distance_steps; step++){
				//To adjust for last iteration!
				int num_copy = (max_cached <= numClusters - step*max_cached) ? max_cached : numClusters - step*max_cached;
				printf("num_cpy = %d\n", num_copy);
				hipMemcpyToSymbol(HIP_SYMBOL(d_cons_centers), h_centers_new + step*max_cached*clusterDimension
					, clusterDimension*num_copy*sizeof(float));
				hipDeviceSynchronize();
				calc_distance1 << <NumBlocks, ThreadsPerBlock, smem_size >> >(d_points, d_clusterIdx,
					d_mindistances, step, num_copy, max_cached);
				hipDeviceSynchronize();
			}
			clk2 = microtime();
#else
			for (int step = 0; step < distance_steps; step++){
				//To adjust for last iteration!
				int num_copy = (max_cached <= numClusters - step*max_cached) ? max_cached : numClusters - step*max_cached;
				printf("num_cpy = %d\n", num_copy);
				hipMemcpyToSymbol(HIP_SYMBOL(d_cons_centers), h_centers_new + step*max_cached*clusterDimension
					, clusterDimension*num_copy*sizeof(float));
				hipDeviceSynchronize();
				calc_distance2 << <(int)ceil(numPoints / 1024.0), 1024 >> >(d_points, d_clusterIdx,
					d_mindistances, step, num_copy, max_cached);
				hipDeviceSynchronize();
			}
			clk2 = microtime();
#endif


			printf("PART 2 :Iteration = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

			clk1 = microtime();
			hipMemcpy(d_centers, h_centers_zero, clusterDimension*numClusters*sizeof(float), hipMemcpyHostToDevice);
#if (48 * 1024) / (numClusters * 4) >= 5
			generate_new_center1 << <NumBlocks, ThreadsPerBlock, smem_size >> >(d_points, d_centers, d_clusterIdx, d_member_counter, split_steps, split_size);
#else
			generate_new_center0 << <(int)ceil(numPoints / 1024.0), 1024 >> >(d_points, d_centers, d_clusterIdx, d_member_counter);
#endif

			hipDeviceSynchronize();
			clk2 = microtime();
			printf("PART 3 :Iteration = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

			clk1 = microtime();
			hipMemcpy(h_centers_new, d_centers, clusterDimension*numClusters*sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_member_counter, d_member_counter, numClusters*sizeof(int), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			clk2 = microtime();
			printf("PART 4 :Iteration = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

			clk1 = microtime();
			member_division(h_centers_new, h_member_counter);
			diff_norm = calculate_norm(h_centers_old, h_centers_new);
			copy_centers(h_centers_old, h_centers_new);

			clk2 = microtime();
			printf("PART 5 :Iteration = %d\t Time = %g\t DN = %g \n", count, (double)(clk2 - clk1), diff_norm);
		}

		kmeansclk2 = microtime();
		printf("Kmeans Total Time = %g seconds\n\n", (double)((kmeansclk2 - kmeansclk1) / 1000000));

		hipMemcpy(h_mindistances, d_mindistances, numPoints*sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();

		for (int i = 0; i < numPoints; i++){
			RSS_score += h_mindistances[i];
		}
		printf("Iteration = %d\t RSS_score is : %f\n", count, RSS_score);

		if (RSS_score < minRSS_score){
			/*Storing back the clusters which exhibit the lowest RSS_score*/
			minRSS_score = RSS_score;
			hipMemcpy(h_min_RSS_clusterIdx, d_clusterIdx, numPoints*sizeof(int), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();

		}
		RSS_score = 0.0;
		diff_norm = (float)FLOAT_MAX;
		count++;
	}


	FILE * fp;
	fp = fopen("clusters.txt", "w");
	fprintf(fp, "RSS_score = %g\n", minRSS_score);
	for (int i = 0; i < numPoints; i++){
		fprintf(fp, "%d\t%d\n", i, h_min_RSS_clusterIdx[i]);
	}
	fclose(fp);
	printf("Done\n");
	hipFree(d_member_counter);
	hipFree(d_clusterIdx);
	hipFree(d_points);
	hipFree(d_centers);
	free(h_points);
	free(h_centers_old);
	free(h_centers_new);
	free(h_clusterIdx);
	free(h_member_counter);
	mclk2 = microtime();

	printf("Total Time = %g seconds\n", (double)((mclk2 - mclk1) / 1000000));
	return 0;
}
