#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <math.h>
#include <time.h>

#define FLOAT_MAX 1e+37
#define numPoints 1024*1024
#define clusterDimension 16
#define numClusters (1024*3 )
#define ConstantMemFloats (64*1024)/4			//	64KB/4
#define rand_range 100
#define PC 0
#if PC == 0
double microtime() { return time(NULL); }
#else
#include <sys/time.h>
double microtime(void)
{
	struct timeval t;
	gettimeofday(&t, 0);
	return 1.0e6*t.tv_sec + (double)t.tv_usec;
}
#endif



__constant__ float d_cons_centers[ConstantMemFloats];

__host__ void generate_random_data(float *h_points, float *h_centers_old, float *h_centers_new){

	//Randomly generating points using rand()
	srand((unsigned int)time(0));
	for (int i = 0; i < numPoints; i++){
		for (int j = 0; j < clusterDimension; j++)
		{
			h_points[i*clusterDimension + j] = (float)(rand() % rand_range);
		}
	}

	//Selecting the first numClusters points as the starting centers
	int k = 0;
	for (int i = 0; i < numClusters; i++, k++){
		for (int j = 0; j < clusterDimension; j++)
		{
			h_centers_old[i*clusterDimension + j] = h_points[k*clusterDimension + j];
			h_centers_new[i*clusterDimension + j] = h_points[k*clusterDimension + j];
		}
	}

}

__host__ float calculate_norm(float *h_centers_old, float *h_centers_new){
	float diff_norm = 0.0f;
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			diff_norm += fabsf(h_centers_old[i*clusterDimension + j] - h_centers_new[i*clusterDimension + j]);
		}
	}

	return diff_norm;
}

__host__ void member_division(float *h_centers_new, int *h_memberCounter){
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			h_centers_new[i*clusterDimension + j] /= h_memberCounter[i];
		}
	}

}

__host__ void copy_centers(float * h_centers_old, float* h_centers_new){
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			h_centers_old[i*clusterDimension + j] = h_centers_new[i*clusterDimension + j];

		}
	}
}



//Costs 500 µ seconds! do not use!
__device__ float distance_func(float *point1, float *point2){
	float distance = 0.0f;
	for (int k = 0; k < clusterDimension; k++){
		distance += sqrtf((point1[k] - point2[k]) * (point1[k] - point2[k]));
	}
	return distance;

}

__global__ void calc_distance(float *d_points, int *d_clusterIdx, float *d_mindistances, int step, int num_copy, int max_cached){

	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int tx = threadIdx.x;
	int min_pos = -1;
	extern __shared__ float s_points[];

	/*Getting the value from previous iterations*/
	if (i < numPoints){
		float min_dist = d_mindistances[i];
		float old_min_dist = d_mindistances[i];

		for (int j = 0; j < clusterDimension; j++){
			s_points[tx*clusterDimension + j] = d_points[i*clusterDimension + j];
		}

		for (int k = 0; k < num_copy; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(s_points[tx*clusterDimension + j] - d_cons_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}

		/*Only update if there were changes!!*/
		if (min_dist < old_min_dist){
			d_mindistances[i] = min_dist;
			d_clusterIdx[i] = step*max_cached + min_pos;
		}
	}
}

__global__ void generate_new_center(float *d_points, float *d_centers, int *d_clusterIdx, int * d_member_counter){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		for (int j = 0; j < clusterDimension; j++){
			atomicAdd(&d_centers[clusterDimension*clusterId + j], d_points[i*clusterDimension + j]);
		}
		atomicAdd(&d_member_counter[clusterId], 1);
	}

}

__global__ void generate_new_center2(float *d_points, float *d_centers, int *d_clusterIdx, int *d_member_counter, int split_steps, int split_size){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int tx = threadIdx.x;
	extern __shared__ float s_centers[];
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[i*clusterDimension + j];
		}


		for (int j = 0; j < split_steps; j++){
			/*
			0*3 -> 0 to 2
			1*3 -> 3 to 5
			2*3 -> 6 to 8
			3*3 -> 9 to 11
			4*3 -> 12 to 14
			5*3 -> 15 to 17
			end   -> 15 , 16 , 17 , 18
			start -> 13 , 14 , 15 , 16
			diff  ->  3  , 2  , 1  , 0
			*/
			__syncthreads();
			int max_length;
			int end_point = (j + 1)*split_size - 1;
			if (end_point <= clusterDimension - 1){
				max_length = split_size;
			}
			else{
				max_length = clusterDimension - j*split_size;
			}

			/*Collabaratively Load 0's in shared memory!*/
			/*Each thread 0's out ele_to_zero entries*/
			int ele_to_zero = ceil((48 * 1024) / (4.0 * blockDim.x));
			if (tx == 0){
				for (int k = 0; k < 48 * 1024 / 4; k++){
					s_centers[k] = 0;
				}
			}



			__syncthreads();

			for (int k = 0; k < max_length; k++){
				atomicAdd(&s_centers[max_length*clusterId + k], points[k]);
			}
			__syncthreads();

			if (tx == 0){
				for (int l = 0; l < numClusters; l++){
					for (int k = 0; k < max_length; k++){
						atomicAdd(&d_centers[j*split_size + clusterDimension*l + k], s_centers[max_length*l + k]);
					}
				}
			}

			__syncthreads();

		}

		atomicAdd(&d_member_counter[clusterId], 1);
	}
}

__host__ void printDeviceInfo(){
	FILE * fp;
	fp = fopen("specifications.txt", "w");
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		fprintf(fp, "Name = %s\n", prop.name);
		fprintf(fp, "Global Memory = %u B\n", (int)prop.totalGlobalMem);
		fprintf(fp, "Shared memory per SM = %d B\n", (int)prop.sharedMemPerBlock);
		fprintf(fp, "Registers per SM = %d\n", (int)prop.regsPerBlock);
		fprintf(fp, "Warp size = %d threads\n", (int)prop.warpSize);
		fprintf(fp, "Max threads per Block = %d\n", (int)prop.maxThreadsPerBlock);
		fprintf(fp, "Total Constant Memory = %d B\n", (int)prop.totalConstMem);

		fprintf(fp, "Texture alignment = %d\n", (int)prop.textureAlignment);
		fprintf(fp, "Device overlap = %d\n", (int)prop.deviceOverlap);
		fprintf(fp, "Number of Multi-processors = %d\n", (int)prop.multiProcessorCount);
		fprintf(fp, "L2 SIZE = %d bytes\n", (int)prop.l2CacheSize);
		fprintf(fp, "Bus Width = %d bits\n", (int)prop.memoryBusWidth);
	}


}

int main(int argc, char **argv){

	printDeviceInfo();
	double clk1, clk2, mclk1, mclk2;
	mclk1 = microtime();


	float *d_points, *d_centers, *d_mindistances;
	int *d_clusterIdx, *d_member_counter;
	float diff_norm = (float)FLOAT_MAX;

	//Calculations for calc_distance()
	/*Block fitting*/
	int ThreadsPerBlock = (48 * 1024) / (clusterDimension*sizeof(float));
	int NumBlocks = (int)ceil(numPoints*1.0f / ThreadsPerBlock);

	/*Shared memory max fitting*/
	int smem_size = ThreadsPerBlock*clusterDimension*sizeof(float);

	/*Constant memory max fitting*/
	int max_cached, distance_steps;
	if (numClusters*clusterDimension * 4.0 <= 64 * 1024 * 1.0f){
		/*I can fit all the centers!*/
		max_cached = numClusters;
		distance_steps = 1;
	}
	else{
		/*One Point takes clusterDimension*4 memory , how mant can I fit in 64k?*/
		max_cached = (int)floor(64 * 1024 * 1.0f / (clusterDimension * 4));
		distance_steps = (int)ceil(numClusters*1.0f / max_cached*1.0f);
	}
	printf("Calculations for max_distance() function!\n");
	printf("numClusters = %d \tThreadsPerBlock = %d \t NumBlocks = %d\t smem_size = %d\n", numClusters, ThreadsPerBlock, NumBlocks, smem_size);
	printf("max_cached %d\t distance_steps %d\n\n", max_cached, distance_steps);

	//Calculations for generate_new_centers()
	int split_size = (int)floor((48 * 1024) / (numClusters * 4.0f));
	int split_steps = (int)ceil(clusterDimension *1.0f / split_size);
	printf("Calculations for generate_new_centers() function!\n");
	printf("split_size = %d \t split_steps = %d \n\n", split_size, split_steps);

	/*Stores the points*/
	float *h_points = (float *)malloc(clusterDimension*numPoints*sizeof(float));

	/*Need two arrays one for old centers, and one for new  for calculating NORM*/
	float *h_centers_old = (float *)malloc(clusterDimension*numClusters*sizeof(float));
	float *h_centers_new = (float *)malloc(clusterDimension*numClusters*sizeof(float));
	float *h_centers_zero = (float *)malloc(clusterDimension*numClusters*sizeof(float));

	/*Stores cluster indexes of all the points*/
	float *h_mindistances = (float *)malloc(numPoints*sizeof(float));
	int *h_clusterIdx = (int *)malloc(numPoints*sizeof(int));

	/*Consists number of members in a clusters*/
	int *h_member_counter = (int *)malloc(numClusters*sizeof(int));

	hipMalloc((void**)& d_points, clusterDimension*numPoints*sizeof(float));
	hipMalloc((void**)& d_centers, clusterDimension*numClusters*sizeof(float));
	hipMalloc((void**)& d_mindistances, numPoints*sizeof(float));
	hipMalloc((void **)&d_clusterIdx, numPoints*sizeof(int));
	hipMalloc((void **)&d_member_counter, numClusters*sizeof(int));
	generate_random_data(h_points, h_centers_old, h_centers_new);


	hipMemcpy(d_points, h_points, clusterDimension*numPoints*sizeof(float), hipMemcpyHostToDevice);
	memset(h_centers_zero, 0, clusterDimension*numClusters*sizeof(float));
	hipDeviceSynchronize();


	int count = 0;
	/*Each co-ordinate has a change less than 0.001 on average!*/
	while (diff_norm > (numPoints*clusterDimension) / 1000.0){
		//while (count < 2){
		clk1 = microtime();

		for (int i = 0; i < numPoints; i++){
			h_clusterIdx[i] = INT_MAX;
			h_mindistances[i] = (float)FLOAT_MAX;
		}
		memset(h_member_counter, 0, numClusters*sizeof(int));
		hipMemcpy(d_clusterIdx, h_clusterIdx, numPoints*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_mindistances, h_mindistances, numPoints*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_centers, h_centers_zero, clusterDimension*numClusters*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_member_counter, h_member_counter, numClusters*sizeof(int), hipMemcpyHostToDevice);

		hipDeviceSynchronize();
		clk2 = microtime();
		printf("PART 1 :Count = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

		clk1 = microtime();

		for (int step = 0; step < distance_steps; step++){
			//To adjust for last iteration!
			int num_copy = (max_cached <= numClusters - step*max_cached) ? max_cached : numClusters - step*max_cached;
			printf("num_cpy = %d\n", num_copy);
			hipMemcpyToSymbol(HIP_SYMBOL(d_cons_centers), h_centers_new + step*max_cached*clusterDimension
				, clusterDimension*num_copy*sizeof(float));
			hipDeviceSynchronize();
			calc_distance << <NumBlocks, ThreadsPerBlock, smem_size >> >(d_points, d_clusterIdx,
				d_mindistances, step, num_copy, max_cached);
			hipDeviceSynchronize();
			clk2 = microtime();

		}
		printf("PART 2 :Count = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

		clk1 = microtime();
		//generate_new_center << <NumBlocks, ThreadsPerBlock >> >(d_points, d_centers, d_clusterIdx, d_member_counter);
		//hipDeviceSynchronize();
		generate_new_center2 << <NumBlocks, ThreadsPerBlock, smem_size >> >(d_points, d_centers, d_clusterIdx, d_member_counter, split_steps, split_size);

		hipDeviceSynchronize();
		clk2 = microtime();
		printf("PART 3 :Count = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

		clk1 = microtime();
		hipMemcpy(h_centers_new, d_centers, clusterDimension*numClusters*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_member_counter, d_member_counter, numClusters*sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		clk2 = microtime();
		printf("PART 4 :Count = %d\t Time = %g µs\n", count, (double)(clk2 - clk1));

		clk1 = microtime();
		member_division(h_centers_new, h_member_counter);
		diff_norm = calculate_norm(h_centers_old, h_centers_new);
		copy_centers(h_centers_old, h_centers_new);

		clk2 = microtime();
		printf("PART 5 :Count = %d\t Time = %g\t DN = %g \n", count, (double)(clk2 - clk1), diff_norm);
	}


	hipMemcpy(h_clusterIdx, d_clusterIdx, numPoints*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for (int i = 0; i < numClusters; i++){
		//fprintf(stderr, "%d\t", h_clusterIdx[i]);
	}

	printf("Done\n");
	hipFree(d_member_counter);
	hipFree(d_clusterIdx);
	hipFree(d_points);
	hipFree(d_centers);
	free(h_points);
	free(h_centers_old);
	free(h_centers_new);
	free(h_clusterIdx);
	free(h_member_counter);
	mclk2 = microtime();

	printf("Total Time = %g seconds\n", (double)((mclk2 - mclk1) / 1000000));
	return 0;
}
