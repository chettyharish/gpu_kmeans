#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <math.h>
#include <time.h>

#define iterations 10						// Max times to run kmeans
#define convergecount 20					// Max kmeans step to avoid flip-flops
#define clusterDimension 16
#define numPoints (1*1024*1024)
#define numClusters (3*1024)
#define FLOAT_MAX 1e+37
#define ConstantMemFloats (64*1024)/4			//	64KB/4
#define SharedMemFloats (24*1024)/4				//	24KB/4 
#define rand_range 100


#define PC 1
#if PC == 1
/*Works on Windows!*/
double microtime() { return (double)time(NULL); }
#else
/*Woks on Linux*/
#include <sys/time.h>
double microtime(void)
{
	struct timeval t;
	gettimeofday(&t, 0);
	return 1.0e6*t.tv_sec + (double)t.tv_usec;
}
#endif


__constant__ float d_cons_centers[ConstantMemFloats];

////////////////////////////////////////////////////////////////////////
////////////////////////UNUSED CODE/////////////////////////////////////
////////////////////////////////////////////////////////////////////////

__host__ void generate_random_points_unused(float *h_points){

	//Randomly generating points using rand()
	srand((unsigned int)time(0));
	int count = 0;
	for (int i = 0; i < numPoints; i++){
		for (int j = 0; j < clusterDimension; j++)
		{
			h_points[i*clusterDimension + j] = (float)(count++);
		}
	}

}


__host__ void generate_random_centers_unused(float *h_points, float *h_centers_old, float *h_centers_new){
	//Selecting random points using Floyd's Algorithm
	int *rand_idx = (int *)malloc(numClusters*sizeof(int));
	int *flag = (int *)malloc(numPoints*sizeof(int));

	memset(rand_idx, 0, numClusters*sizeof(int));
	memset(flag, 0, numPoints*sizeof(int));

	int n = numPoints;
	int m = 0;
	for (n = (numPoints - numClusters); n < numPoints && m < numClusters; n++, m++){
		int r = rand() % (n + 1);

		if (flag[r] == 1){
			/*Works since previous iteration had rand() % n , and thus
			it is not possible that n was chosen!*/
			r = n;
		}
		rand_idx[m] = r;
		flag[r] = 1;
	}


	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++)
		{
			h_centers_old[i*clusterDimension + j] = h_points[rand_idx[i] * clusterDimension + j];
			h_centers_new[i*clusterDimension + j] = h_points[rand_idx[i] * clusterDimension + j];
		}
	}
}

//Costs 500 µ seconds! do not use!
__device__ float distance_func(float *point1, float *point2){
	float distance = 0.0f;
	for (int k = 0; k < clusterDimension; k++){
		distance += sqrtf((point1[k] - point2[k]) * (point1[k] - point2[k]));
	}
	return distance;

}

/*Points in registers and getting the centers from global memory*/
__global__ void calc_distance0_unused(float *d_centers, float *d_points, int *d_clusterIdx, float *d_mindistances){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int min_pos = -1;
	if (i < numPoints){
		float min_dist = d_mindistances[i];
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[i*clusterDimension + j];
		}


		for (int k = 0; k < numClusters; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(points[j] - d_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}

		d_mindistances[i] = min_dist;
		d_clusterIdx[i] = min_pos;
	}
}

/*Points in shared memory and centers in constant memory*/
__global__ void calc_distance1_unused(float *d_points, int *d_clusterIdx, float *d_mindistances, int step, int num_copy, int max_cached){

	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int tx = threadIdx.x;
	int min_pos = -1;
	extern __shared__ float s_points[];

	/*Getting the value from previous iterations*/
	if (i < numPoints){
		float min_dist = d_mindistances[i];
		float old_min_dist = d_mindistances[i];

		for (int j = 0; j < clusterDimension; j++){
			s_points[tx*clusterDimension + j] = d_points[i*clusterDimension + j];
		}

		for (int k = 0; k < num_copy; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(s_points[tx*clusterDimension + j] - d_cons_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}

		/*Only update if there were changes!!*/
		if (min_dist < old_min_dist){
			d_mindistances[i] = min_dist;
			d_clusterIdx[i] = step*max_cached + min_pos;
		}
	}
}


/*
Points in registers and centers in constant memory
*/
__global__ void calc_distance2_unused(float *d_points, int *d_clusterIdx, float *d_mindistances, int step, int num_copy, int max_cached){

	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int min_pos = -1;

	/*Getting the value from previous iterations*/
	if (i < numPoints){
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[i*clusterDimension + j];
		}
		float min_dist = d_mindistances[i];
		float old_min_dist = d_mindistances[i];

		for (int k = 0; k < num_copy; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(points[j] - d_cons_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}

		/*Only update if there were changes!!*/
		if (min_dist < old_min_dist){
			d_mindistances[i] = min_dist;
			d_clusterIdx[i] = step*max_cached + min_pos;
		}
	}
}

/*Trying to read from adjacent locations in constant memory
Using magic index as a stride aligner*/
__global__ void calc_distance_transpose_unused(float *d_points, int *d_clusterIdx, float *d_mindistances, int step, int num_copy, int max_cached){

	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int tx = threadIdx.x;
	int min_pos = -1;

	/*Getting the value from previous iterations*/
	if (i < numPoints){
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[j*numPoints + i];
		}
		float min_dist = d_mindistances[i];
		float old_min_dist = d_mindistances[i];

		/*Forcing adjacent accesses by number magic :)*/
		for (int k = 0; k < num_copy; k++){
			int magic = (tx + k) % num_copy;
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(points[j] - d_cons_centers[j*num_copy + magic]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = magic;
			}
		}

		/*Only update if there were changes!!*/
		if (min_dist < old_min_dist){
			d_mindistances[i] = min_dist;
			d_clusterIdx[i] = step*max_cached + min_pos;
		}
	}
}


__global__ void generate_new_center_transpose_unused(float *d_points, float *d_centers, int *d_clusterIdx, int * d_member_counter){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		for (int j = 0; j < clusterDimension; j++){
			atomicAdd(&d_centers[clusterDimension*clusterId + j], d_points[j*numPoints + i]);
		}
		atomicAdd(&d_member_counter[clusterId], 1);
	}

}

__global__ void generate_new_center2_unused(float *d_points, float *d_centers, int *d_clusterIdx, int *d_member_counter, int split_steps, int split_size){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int tx = threadIdx.x;
	extern __shared__ float s_centers[];
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[i*clusterDimension + j];
		}


		for (int j = 0; j < split_steps; j++){
			/*
			0*3 -> 0 to 2
			1*3 -> 3 to 5
			2*3 -> 6 to 8
			3*3 -> 9 to 11
			4*3 -> 12 to 14
			5*3 -> 15 to 17
			end   -> 15 , 16 , 17 , 18
			start -> 13 , 14 , 15 , 16
			diff  ->  3  , 2  , 1  , 0
			*/
			__syncthreads();
			int max_length;
			int end_point = (j + 1)*split_size - 1;
			if (end_point <= clusterDimension - 1){
				max_length = split_size;
			}
			else{
				max_length = clusterDimension - j*split_size;
			}

			/*Collabaratively Load 0's in shared memory!*/
			/*Each thread 0's out ele_to_zero entries*/
			int ele_to_zero = ceil((48 * 1024) / (4.0 * blockDim.x));
			if (tx == 0){
				for (int k = 0; k < 48 * 1024 / 4; k++){
					s_centers[k] = 0;
				}
			}



			__syncthreads();

			for (int k = 0; k < max_length; k++){
				atomicAdd(&s_centers[max_length*clusterId + k], points[k]);
			}
			__syncthreads();

			if (tx == 0){
				for (int l = 0; l < numClusters; l++){
					for (int k = 0; k < max_length; k++){
						atomicAdd(&d_centers[j*split_size + clusterDimension*l + k], s_centers[max_length*l + k]);
					}
				}
			}

			__syncthreads();

		}

		atomicAdd(&d_member_counter[clusterId], 1);
	}
}

////////////////////////////////////////////////////////////////////////
////////////////////////UNUSED CODE ENDS///////////////////////////////
////////////////////////////////////////////////////////////////////////


__host__ void printDeviceInfo(){
	FILE * fp;
	fp = fopen("specifications.txt", "w");
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		fprintf(fp, "Name = %s\n", prop.name);
		fprintf(fp, "Number of Multi-processors = %d\n", (int)prop.multiProcessorCount);
		fprintf(fp, "Max threads per Block = %d\n", (int)prop.maxThreadsPerBlock);
		fprintf(fp, "Max threads per SM = %d\n", (int)prop.maxThreadsPerMultiProcessor);

		fprintf(fp, "Global Memory = %u B\n", (int)prop.totalGlobalMem);
		fprintf(fp, "L2 SIZE = %d bytes\n", (int)prop.l2CacheSize);
		fprintf(fp, "Shared memory per SM = %d B\n", (int)prop.sharedMemPerBlock);
		fprintf(fp, "Total Constant Memory = %d B\n", (int)prop.totalConstMem);

		fprintf(fp, "Registers per SM = %d\n", (int)prop.regsPerBlock);
		fprintf(fp, "Average Registers per Thread = %d\n", (int)prop.regsPerBlock / (int)prop.maxThreadsPerMultiProcessor);
		fprintf(fp, "Warp size = %d threads\n", (int)prop.warpSize);


		fprintf(fp, "Version = %d.%d\n", (int)prop.major, (int)prop.minor);
	}

	fclose(fp);
}

__host__ void create_input_file(){
	/*Writes the input to a file for testing with other codes*/
	int count = 0;
	FILE * fp;
	fp = fopen("data.txt", "w");
	for (int i = 0; i < numPoints; i++)
	{
		fprintf(fp, "%d ", i);
		for (int j = 0; j < clusterDimension; j++)
		{
			if (j != clusterDimension - 1)
				fprintf(fp, "%d ", count);
			else
				fprintf(fp, "%d\n", count);
			count++;
		}

	}
}

__host__ void generate_random_points_transpose(float *h_points){
	/*Avoiding rand() since its not truly random*/
	//Randomly generating points using rand()
	//srand((unsigned int)time(0));

	int count = 0;
	for (int j = 0; j < clusterDimension; j++)
	{
		for (int i = 0; i < numPoints; i++){
			//h_points[j*numPoints + i] = (float)(rand() % rand_range);
			h_points[j*numPoints + i] = (float)(count++);
		}
	}
}

__host__ void generate_random_centers_transpose(float *h_points, float *h_centers_old, float *h_centers_new){
	//Selecting random points using Floyd's Algorithm
	int *rand_idx = (int *)malloc(numClusters*sizeof(int));
	int *flag = (int *)malloc(numPoints*sizeof(int));

	memset(rand_idx, 0, numClusters*sizeof(int));
	memset(flag, 0, numPoints*sizeof(int));

	int n = numPoints;
	int m = 0;
	for (n = (numPoints - numClusters); n < numPoints && m < numClusters; n++, m++){
		int r = rand() % (n + 1);

		if (flag[r] == 1){
			/*Works since previous iteration had rand() % n , and thus
			it is not possible that n was chosen!*/
			r = n;
		}
		rand_idx[m] = r;
		flag[r] = 1;
	}


	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++)
		{
			h_centers_old[i*clusterDimension + j] = h_points[j*numPoints + rand_idx[i]];
			h_centers_new[i*clusterDimension + j] = h_points[j*numPoints + rand_idx[i]];
		}
	}

}



__host__ float calculate_norm(float *h_centers_old, float *h_centers_new){
	float diff_norm = 0.0f;
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			diff_norm += fabsf(h_centers_old[i*clusterDimension + j] - h_centers_new[i*clusterDimension + j]);
		}
	}

	return diff_norm;
}

__host__ void member_division(float *h_centers_new, int *h_memberCounter){
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			/*New center is same as old if no points in the cluster*/
			if (h_memberCounter[i] != 0)
				h_centers_new[i*clusterDimension + j] /= h_memberCounter[i];
		}
	}
}

__host__ void copy_centers(float * h_centers_old, float* h_centers_new){
	for (int i = 0; i < numClusters; i++){
		for (int j = 0; j < clusterDimension; j++){
			h_centers_old[i*clusterDimension + j] = h_centers_new[i*clusterDimension + j];

		}
	}
}


/*
A modified kernel which tries to take advantage of both the orders
Uses column ordering for points and row ordering for centers
*/

__global__ void calc_distance_constant(float *d_points, int *d_clusterIdx, float *d_mindistances, int step_constant, int num_copy_constant, int max_cached_constant){

	int i = blockDim.x*blockIdx.x + threadIdx.x;

	/*Getting the value from previous iterations*/
	if (i < numPoints){
		int min_pos = -1;
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[j*numPoints + i];
		}
		float min_dist = d_mindistances[i];
		float old_min_dist = d_mindistances[i];

		for (int k = 0; k < num_copy_constant; k++){
			float distance = 0.0f;
			for (int j = 0; j < clusterDimension; j++){
				distance += fabsf(points[j] - d_cons_centers[k*clusterDimension + j]);
			}

			if (distance < min_dist){
				min_dist = distance;
				min_pos = k;
			}
		}


		/*Only update if there were changes!!*/
		if (min_dist < old_min_dist){
			d_mindistances[i] = min_dist;
			d_clusterIdx[i] = step_constant*max_cached_constant + min_pos;
		}
	}
}


__global__ void generate_new_center_transpose(float *d_points, float *d_centers, int *d_clusterIdx, int * d_member_counter){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		float points[clusterDimension];
		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[j*numPoints + i];
		}
		for (int j = 0; j < clusterDimension; j++){
			atomicAdd(&d_centers[clusterDimension*clusterId + j], points[j]);
		}
		atomicAdd(&d_member_counter[clusterId], 1);
	}

}

__global__ void add_member_counts(int *d_clusterIdx, int * d_member_counter){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	extern __shared__ float s_member_counter[];
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		int tx = threadIdx.x;

		int member_counts_to_load = numClusters / 1024.0;
		for (int l = tx; l < tx + member_counts_to_load; l++){
			s_member_counter[l] = 0;
		}
		__syncthreads();
		atomicAdd(&s_member_counter[clusterId], 1);
		__syncthreads();

		for (int l = tx; l < tx + member_counts_to_load; l++){
			atomicAdd(&d_member_counter[l], s_member_counter[l]);
		}
	}
}

__global__ void generate_new_center_shared(float *d_points, float *d_centers, int *d_clusterIdx, int *d_member_counter, int split_steps, int split_size){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	extern __shared__ float s_centers[];
	if (i < numPoints){
		int clusterId = d_clusterIdx[i];
		int tx = threadIdx.x;
		float points[clusterDimension];

		for (int j = 0; j < clusterDimension; j++){
			points[j] = d_points[j*numPoints + i];
		}


		for (int j = 0; j < split_steps; j++){
			__syncthreads();
			int max_length;
			int end_point = (j + 1)*split_size - 1;
			if (end_point <= clusterDimension - 1){
				max_length = split_size;
			}
			else{
				max_length = clusterDimension - j*split_size;
			}

			/*Collabaratively Load 0's in shared memory!*/
			/*We know it has (24*1024/4)) floats*/
			/*So each thread has to zero out 6 elements*/
			for (int k = 0; k < 6; k++){
				s_centers[6 * tx + k] = 0;
			}
			__syncthreads();

			for (int k = 0; k < max_length; k++){
				atomicAdd(&s_centers[max_length*clusterId + k], points[j*split_size + k]);
			}
			__syncthreads();



			/*Threads must collabaratively writeback to d_centers , where each thread takes out one center*/
			/*Each thread must writeback max_length elements to d_centers*/
			/*Threads must only writeback 24K in total!!! must be careful of centers < threads*/
			/*Number of centers will be greater than number of threads!*/

			int num_cluster_to_writeback = numClusters / 1024.0;
			for (int l = tx; l < tx + num_cluster_to_writeback; l++){
				for (int k = 0; k < max_length; k++){
					atomicAdd(&d_centers[j*split_size + clusterDimension*l + k], s_centers[max_length*l + k]);
				}
			}
			__syncthreads();

		}

		atomicAdd(&d_member_counter[clusterId], 1);
	}
}

__global__ void add_array(float *d_mindistances)
{
	const int threads_per_block = 1024;
	__shared__ float partialSum[2 * threads_per_block];
	int idx = threadIdx.x;
	int start = 2 * blockIdx.x * threads_per_block;
	if (start < numPoints){
		partialSum[idx] = d_mindistances[start + idx];
		if (start + threads_per_block < numPoints)
			partialSum[idx + threads_per_block] = d_mindistances[start + idx + threads_per_block];
		else
			partialSum[idx + threads_per_block] = 0;

		for (unsigned int stride = threads_per_block; stride >= 1; stride >>= 1)
		{
			__syncthreads();
			if (idx < stride)
				partialSum[idx] += partialSum[idx + stride];
		}

		d_mindistances[start + idx] = partialSum[idx];
	}
}

int main(int argc, char **argv){

	printDeviceInfo();
	create_input_file();
	exit(1);
	double clk1, clk2, mclk1, mclk2, kmeansclk1, kmeansclk2;
	mclk1 = microtime();

	/*For maintaining multiple iterations of kmeans*/
	/*Usese RSS(Residual sum of squares) to determine best iteration*/
	/*RSS_score = sum((di)^2) for all di , where di is distance from cluster center*/
	double RSS_score = 0.0;
	double minRSS_score = (float)FLOAT_MAX;
	int *h_min_RSS_clusterIdx = (int *)malloc(numPoints*sizeof(int));

	/*For carrying out one iteration of Kmeans*/
	float *d_points, *d_centers, *d_mindistances;
	int *d_clusterIdx, *d_member_counter;
	float diff_norm = (float)FLOAT_MAX;


	/*Shared memory max fitting , Ensuring 2 blocks fitting at a time*/
	int smem_size = 24 * 1024;


	/*Constant memory max fitting*/
	int max_cached_constant, distance_steps_constant;
	if (numClusters*clusterDimension * 4.0 <= 64 * 1024 * 1.0f){
		/*I can fit all the centers!*/
		max_cached_constant = numClusters;
	}
	else{
		/*One Point takes clusterDimension*4 memory , how mant can I fit in 64k?*/
		max_cached_constant = (int)floor(64 * 1024 * 1.0f / (clusterDimension * 4));
	}

	distance_steps_constant = (int)ceil(numClusters*1.0f / max_cached_constant*1.0f);
	printf("Calculations for max_distance() function!\n");
	printf("smem_size = %d\t max_cached %d\t distance_steps %d\n\n", smem_size, max_cached_constant, distance_steps_constant);

	//Calculations for generate_new_centers()
	int split_size = (int)floor((24 * 1024) / (numClusters * 4.0f));
	int split_steps = (int)ceil(clusterDimension *1.0f / split_size);
	printf("Calculations for generate_new_centers() function!\n");
	printf("split_size = %d \t split_steps = %d \n\n", split_size, split_steps);


	/*Stores the points*/
	float *h_points = (float *)malloc(clusterDimension*numPoints*sizeof(float));

	/*Need two arrays one for old centers, and one for new  for calculating NORM*/
	float *h_centers_old = (float *)malloc(clusterDimension*numClusters*sizeof(float));
	float *h_centers_new = (float *)malloc(clusterDimension*numClusters*sizeof(float));
	float *h_centers_transpose = (float *)malloc(clusterDimension*max_cached_constant*sizeof(float));

	/*Stores cluster indexes of all the points*/
	float *h_mindistances = (float *)malloc(numPoints*sizeof(float));
	int *h_clusterIdx = (int *)malloc(numPoints*sizeof(int));

	/*Consists number of members in a clusters*/
	int *h_member_counter = (int *)malloc(numClusters*sizeof(int));

	/*Changing preference to L1Cache for both the kernels*/
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(calc_distance_constant), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(generate_new_center_transpose), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(generate_new_center_shared), hipFuncCachePreferShared);

	hipMalloc((void**)& d_points, clusterDimension*numPoints*sizeof(float));
	hipMalloc((void**)& d_centers, clusterDimension*numClusters*sizeof(float));
	hipMalloc((void**)& d_mindistances, numPoints*sizeof(float));
	hipMalloc((void **)&d_clusterIdx, numPoints*sizeof(int));
	hipMalloc((void **)&d_member_counter, numClusters*sizeof(int));
	generate_random_points_transpose(h_points);


	hipMemcpy(d_points, h_points, clusterDimension*numPoints*sizeof(float), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	int count = 0;
	int con_count = 0;
	while (count < iterations){

		generate_random_centers_transpose(h_points, h_centers_old, h_centers_new);
		/*Each co-ordinate has a change less than 0.001 on average!*/
		kmeansclk1 = microtime();
		while (diff_norm > (numPoints*clusterDimension) / 1000.0 && con_count < convergecount){
			printf("\nIteration = %d \t Ccount = %d\n", count, con_count);
			clk1 = microtime();
			hipMemset(d_clusterIdx, 1, numPoints*sizeof(int));
			hipMemset(d_mindistances, 99, numPoints*sizeof(float));
			hipMemset(d_member_counter, 0, numClusters*sizeof(int));
			hipDeviceSynchronize();
			clk2 = microtime();
			printf("PART 1 :Time = %g µs\n", (double)(clk2 - clk1));

			clk1 = microtime();
			for (int step_constant = 0; step_constant < distance_steps_constant; step_constant++){
				//To adjust for last iteration!
				int num_copy_constant = (max_cached_constant <= numClusters - step_constant*max_cached_constant) ? max_cached_constant : numClusters - step_constant*max_cached_constant;
				hipMemcpyToSymbol(HIP_SYMBOL(d_cons_centers), h_centers_new + step_constant*max_cached_constant*clusterDimension, clusterDimension*num_copy_constant*sizeof(float));
				hipDeviceSynchronize();
				calc_distance_constant << <(int)ceil(numPoints / 1024.0), 1024 >> >(d_points, d_clusterIdx,
					d_mindistances, step_constant, num_copy_constant, max_cached_constant);
				hipDeviceSynchronize();


				//printf("num_cpy = %d\tcalc_distance_mixed: %s\n", num_copy, hipGetErrorString(hipGetLastError()));
			}
			clk2 = microtime();
			printf("PART 2 :Time = %g µs\n", (double)(clk2 - clk1));

			clk1 = microtime();
			hipMemset(d_centers, 0, clusterDimension*numClusters*sizeof(float));
			hipDeviceSynchronize();
			generate_new_center_transpose << <(int)ceil(numPoints / 1024.0), 1024 >> >(d_points, d_centers, d_clusterIdx, d_member_counter);
			//add_member_counts << <(int)ceil(numPoints / 1024.0), 1024, smem_size >> >(d_clusterIdx , d_member_counter);
			//generate_new_center_shared << <(int)ceil(numPoints / 1024.0), 1024, smem_size >> >(d_points, d_centers, d_clusterIdx, d_member_counter, split_steps, split_size);

			hipDeviceSynchronize();
			//printf("generate_new_center_transpose: %s\n", hipGetErrorString(hipGetLastError()));
			clk2 = microtime();
			printf("PART 3 :Time = %g µs\n", (double)(clk2 - clk1));

			clk1 = microtime();
			hipMemcpy(h_centers_new, d_centers, clusterDimension*numClusters*sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_member_counter, d_member_counter, numClusters*sizeof(int), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			clk2 = microtime();
			printf("PART 4 :Time = %g µs\n", (double)(clk2 - clk1));

			clk1 = microtime();

			/*Handling overlapping centers*/
			int zero_count = 0;
			for (int i = 0; i < numClusters; i++){
				if (h_member_counter[i] == 0)
					/*Since the center itself is one member*/
					/*The logic will end up selecting the first overlapping center otherwise*/
					h_member_counter[i]++;
			}
			printf("Zero Count = %d\n", zero_count);


			member_division(h_centers_new, h_member_counter);
			diff_norm = calculate_norm(h_centers_old, h_centers_new);
			copy_centers(h_centers_old, h_centers_new);
			con_count++;
			clk2 = microtime();
			printf("PART 5 :Time = %g µs\n", (double)(clk2 - clk1));
			printf("Diff Norm = %f\n", diff_norm);
		}
		kmeansclk2 = microtime();
		printf("Kmeans Total Time = %g seconds\n\n", (double)((kmeansclk2 - kmeansclk1) / 1000000));

		/*Use the GPU to add array if the number of elements is high enough!*/
#if numPoints > 32*1024*1024
		add_array << <(int)ceil(numPoints / 1024.0), 1024 >> >(d_mindistances);
		hipMemcpy(h_mindistances, d_mindistances, numPoints*sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		for (int i = 0; i < (int)ceil(numPoints / 1024.0); i++){
			RSS_score += h_mindistances[i * 2 * 1024];
		}
#else
		hipMemcpy(h_mindistances, d_mindistances, numPoints*sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		for (int i = 0; i < numPoints; i++){
			RSS_score += h_mindistances[i];
		}
#endif
		printf("Iteration = %d\t RSS_score is : %f\n", count, RSS_score);


		if (RSS_score < minRSS_score){
			/*Storing back the clusters which exhibit the lowest RSS_score*/
			minRSS_score = RSS_score;
			hipMemcpy(h_min_RSS_clusterIdx, d_clusterIdx, numPoints*sizeof(int), hipMemcpyDeviceToHost);
			hipDeviceSynchronize();

		}
		RSS_score = 0.0;
		con_count = 0;
		diff_norm = (float)FLOAT_MAX;
		count++;

		}


	FILE * fp;
	fp = fopen("clusters.txt", "w");
	fprintf(fp, "RSS_score = %g\n", minRSS_score);
	for (int i = 0; i < numPoints; i++){
		fprintf(fp, "%d\t%d\n", i, h_min_RSS_clusterIdx[i]);
	}
	fclose(fp);
	printf("Done\n");
	hipFree(d_member_counter);
	hipFree(d_clusterIdx);
	hipFree(d_points);
	hipFree(d_centers);
	free(h_points);
	free(h_centers_old);
	free(h_centers_new);
	free(h_clusterIdx);
	free(h_member_counter);
	mclk2 = microtime();

	printf("Total Time = %g seconds\n", (double)((mclk2 - mclk1) / 1000000));
	return 0;
}
